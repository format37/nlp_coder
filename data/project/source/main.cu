#include <stdio.h>
#include <hip/hip_runtime.h>
#include "bignum.h"
#include "test_div.h"
#include "c_test.h"

int main() {
    
    BN_ULONG test_values_dividend[][MAX_BIGNUM_WORDS] = {        
        {0,0,0,0xB}, // 0
        {0x1,0,0,0}, // 1
        {0,0,0x1234567890ABCDEF,0x7234567890ABCDEF} // 2
    };

    BN_ULONG test_values_divisor[][MAX_BIGNUM_WORDS] = {        
        {0,0,0,0x3}, // 0
        {0x2,0,0,0}, // 1
        {0,0,0x2,0} // 2
    };

    int num_tests = sizeof(test_values_dividend) / sizeof(test_values_dividend[0]);

    //BN_ULONG A[WORDS] = {0, 0, 0, 0xB};
    //BN_ULONG B[WORDS] = {0, 0, 0, 0x3};
    printf("\n\n### C test, expected results:\n");
    for (int i = 0; i < num_tests; i++) {
        printf("Test %d:\n", i);
        c_bn_div_test(test_values_dividend[i], test_values_divisor[i]);
    }
    // c_bn_div_test(test_values_dividend, test_values_divisor, num_tests);

    /*printf("\n\n### CUDA test:\n");
    // Launch the kernel to run the test
    test_div<<<1, 1>>>();

    // Check for any errors after running the kernel
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
        return -1;
    }

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();*/
    return 0;
}