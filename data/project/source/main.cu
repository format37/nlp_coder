#include <stdio.h>
#include <hip/hip_runtime.h>
#include "bignum.h"
#include "test_div.h"
#include "c_test.h"

int main() {
    BN_ULONG test_values_dividend[][MAX_BIGNUM_WORDS] = {
        {0,0,0,0xB}, // 0
        {0x1,0,0,0}, // 1
        {0,0,0x1234567890ABCDEF,0x7234567890ABCDEF}, // 2
        {0x1,0,0,0} // 3
    };

    BN_ULONG test_values_divisor[][MAX_BIGNUM_WORDS] = {
        {0,0,0,0x3}, // 0
        {0x2,0,0,0}, // 1
        {0,0,0x2,0}, // 2
        {0,0,0x100,0} // 3
    };

    int num_tests = sizeof(test_values_dividend) / sizeof(test_values_dividend[0]);

    printf("\n\n### C test, expected results:\n");
    for (int i = 0; i < num_tests; i++) {
        printf("\nTest %d:\n", i);
        c_bn_div_test(test_values_dividend[i], test_values_divisor[i]);
    }

    printf("\n\n### CUDA test:\n");

    BN_ULONG *d_A, *d_B;
    hipMalloc((void**)&d_A, MAX_BIGNUM_WORDS * sizeof(BN_ULONG));
    hipMalloc((void**)&d_B, MAX_BIGNUM_WORDS * sizeof(BN_ULONG));

    for (int i = 0; i < num_tests; i++) {
        printf("\nTest %d:\n", i);

        hipMemcpy(d_A, test_values_dividend[i], MAX_BIGNUM_WORDS * sizeof(BN_ULONG), hipMemcpyHostToDevice);
        hipMemcpy(d_B, test_values_divisor[i], MAX_BIGNUM_WORDS * sizeof(BN_ULONG), hipMemcpyHostToDevice);

        // Launch the kernel to run the test
        test_div<<<1, 1>>>(d_A, d_B);

        // Check for any errors after running the kernel
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("Error: %s\n", hipGetErrorString(err));
            return -1;
        }

        // Wait for GPU to finish before accessing on host
        hipDeviceSynchronize();
    }

    hipFree(d_A);
    hipFree(d_B);
    return 0;
}