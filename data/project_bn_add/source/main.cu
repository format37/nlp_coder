#include <stdio.h>
#include <hip/hip_runtime.h>
#include "bignum.h"
#include "test_add.h"

int main() {
    // Launch the kernel to run the test
    test_add<<<1, 1>>>();

    // Check for any errors after running the kernel
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
        return -1;
    }

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
    return 0;
}